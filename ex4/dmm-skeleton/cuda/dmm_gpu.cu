#include "hip/hip_runtime.h"
/*
 *  dmm_gpu.cu -- Template for DMM GPU kernels
 *
 *  Copyright (C) 2019, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2019, Athena Elafrou
 */

#include "dmm.h"

/*
 *  Naive kernel
 */
__global__ void dmm_gpu_naive(const value_t *A, const value_t *B, value_t *C,
                              const size_t M, const size_t N, const size_t K) {
  /*
   * FILLME: fill the code.
   */
}

/*
 *  Coalesced memory acceses of A.
 */
__global__ void dmm_gpu_coalesced_A(const value_t *A, const value_t *B,
				    value_t *C, const size_t M, const size_t N,
				    const size_t K) {
  /*
   * FILLME: fill the code.
   */
}

/*
 *  Reduced memory accesses.
 */
__global__ void dmm_gpu_reduced_global(const value_t *A, const value_t *B, value_t *C,
				       const size_t M, const size_t N, const size_t K) {
  /*
   * FILLME: fill the code.
   */
}

/*
 *  Use of cuBLAS
 */
void dmm_gpu_cublas(const value_t *A, const value_t *B, value_t *C,
		    const size_t M, const size_t N, const size_t K) {
  /*
   * FILLME: fill the code.
   */
}
